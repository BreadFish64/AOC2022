#include "hip/hip_runtime.h"
// Does not work

#include "pch.hpp"

#include "cuda.hpp"

struct Interval {
    s32                      lower;
    s32                      upper;
    __host__ __device__ bool empty() const { return lower >= upper; }
};
__host__ __device__ bool Intersect(const Interval& lhs, const Interval& rhs) {
    return lhs.lower <= rhs.upper && rhs.lower <= lhs.upper;
}
__host__ __device__ bool Touch(const Interval& lhs, const Interval& rhs) {
    return Intersect(lhs, {rhs.lower - 1, rhs.upper + 1});
}

using Point = Eigen::Vector<s32, 2>;

struct Sensor {
    Point position{0, 0};
    Point closest_beacon{0, 0};
    s32   beacon_distance{};

    __host__ __device__ void updateBeaconDistance() { beacon_distance = (closest_beacon - position).cwiseAbs().sum(); }
    __host__ __device__ Interval exclusionZoneAt(s32 y) const {
        auto y_diff = std::abs(y - position[1]);
        auto x_diff = beacon_distance - y_diff;
        if (x_diff < 0) return {0,0};
        return {position[0] - x_diff, position[0] + x_diff};
    }
    __host__ __device__ Interval exclusionZoneY() const {
        return {position[1] - beacon_distance, position[1] + beacon_distance};
    }

    static bool LowerX(const Sensor& lhs, const Sensor& rhs) { return lhs.position[0] < rhs.position[0]; }
};

constexpr s32 check_row = 2000000;
constexpr s32 max_row   = 4000000;
constexpr s32 row_count = max_row + 1;

__global__ void UpdateBeacons(Sensor* sensors, u32 sensor_count) {
    u32 index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < sensor_count) sensors[index].updateBeaconDistance();
}

__global__ void Evaluate(Sensor* sensors, u32 sensor_count) {
    s32 y = blockDim.x * blockIdx.x + threadIdx.x;
    if (y > max_row) return;

    Interval intervals[16];
    u32      interval_count = 0;

    for (u32 s = 0; s < sensor_count; ++s) {
        Interval y_ranges = sensors[s].exclusionZoneY();
        if (y < y_ranges.lower || y > y_ranges.upper) continue;
        Interval interval = sensors[s].exclusionZoneAt(y);
        if (interval.lower < 0) interval.lower = 0;
        if (interval.upper > max_row) interval.upper = max_row;
        if (interval.empty()) continue;
        for (u32 i = 0; i < interval_count; ++i) {
            Interval existing_interval = intervals[i];
            if (Touch(interval, existing_interval)) {
                intervals[i] = {
                    std::min(existing_interval.lower, interval.lower),
                    std::max(existing_interval.upper, interval.upper),
                };
                goto END_SENSOR_LOOP;
            }
        }
        intervals[interval_count++] = interval;
    END_SENSOR_LOOP:
    }

    s32 maybe_empty = intervals[0].upper + 1;
    for (u32 i = 1; i < interval_count; ++i) {
        Interval interval = intervals[1];
        if (interval.lower > maybe_empty) {
            sensors[sensor_count].position = {maybe_empty, y};
            return;
        }
        maybe_empty = std::max(maybe_empty, interval.upper + 1);
    }
}

int main() {
    std::array<Sensor, 34> sensors{{
        {{2302110, 2237242}, {2348729, 1239977}}, {{47903, 2473047}, {432198, 2000000}},
        {{2363579, 1547888}, {2348729, 1239977}}, {{3619841, 520506}, {2348729, 1239977}},
        {{3941908, 3526118}, {3772294, 3485243}}, {{3206, 1564595}, {432198, 2000000}},
        {{3123411, 3392077}, {2977835, 3592946}}, {{3279053, 3984688}, {2977835, 3592946}},
        {{2968162, 3938490}, {2977835, 3592946}}, {{1772120, 2862246}, {2017966, 3158243}},
        {{3283241, 2619168}, {3172577, 2521434}}, {{2471642, 3890150}, {2977835, 3592946}},
        {{3163348, 3743489}, {2977835, 3592946}}, {{2933313, 2919047}, {3172577, 2521434}},
        {{2780640, 3629927}, {2977835, 3592946}}, {{3986978, 2079918}, {3998497, 2812428}},
        {{315464, 370694}, {550536, 260566}},     {{3957316, 3968366}, {3772294, 3485243}},
        {{2118533, 1074658}, {2348729, 1239977}}, {{3494855, 3378533}, {3772294, 3485243}},
        {{2575727, 210553}, {2348729, 1239977}},  {{3999990, 2813525}, {3998497, 2812428}},
        {{3658837, 3026912}, {3998497, 2812428}}, {{1551619, 1701155}, {2348729, 1239977}},
        {{2625855, 3330422}, {2977835, 3592946}}, {{3476946, 2445098}, {3172577, 2521434}},
        {{2915568, 1714113}, {2348729, 1239977}}, {{729668, 3723377}, {997494, 3617758}},
        {{3631681, 3801747}, {3772294, 3485243}}, {{2270816, 3197807}, {2017966, 3158243}},
        {{3999999, 2810929}, {3998497, 2812428}}, {{3978805, 3296024}, {3772294, 3485243}},
        {{1054910, 811769}, {2348729, 1239977}},  Sensor{},
    }};
    for (auto& sensor : sensors)
        sensor.updateBeaconDistance();

    auto start = std::chrono::steady_clock::now();
    std::sort(sensors.begin(), sensors.end() - 1, &Sensor::LowerX);
    s64 result{};
    {
        auto sensor_buffer = MakeDeviceBuffer<Sensor>(sensors.size());
        Check(hipMemcpy(sensor_buffer.get(), sensors.data(), sensors.size() * sizeof(Sensor), hipMemcpyHostToDevice));
        //UpdateBeacons<<<DivCeil<u32>(sensors.size(), 256), 256>>>(sensor_buffer.get(), sensors.size() - 1);
        Evaluate<<<DivCeil<u32>(row_count, 256), 256>>>(sensor_buffer.get(), sensors.size() - 1);
        Check(hipMemcpy(sensors.data(), sensor_buffer.get(), sensors.size() * sizeof(Sensor), hipMemcpyDeviceToHost));
        result = static_cast<s64>(sensors.back().position[1]) * max_row + sensors.back().position[0];
    }
    auto stop = std::chrono::steady_clock::now();
    fmt::print("Solve Time: {:.2}\n", std::chrono::duration<double, std::milli>{stop - start});
    fmt::print("Part 2: ({}, {}) -> {}\n", sensors.back().position[0], sensors.back().position[1], result);
}
