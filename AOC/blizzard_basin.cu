#include "hip/hip_runtime.h"
#include "pch.hpp"

#include "cuda.hpp"

std::istringstream test{R"(#.######
#>>.<^<#
#.<..<<#
#>v.><>#
#<^v^^>#
######.#
)"};

using Coord = Eigen::Vector2i;

struct Blizzard {
    Eigen::Vector2i coord;
    char            direction;
};

__global__ void ExpandKernel(hipSurfaceObject_t dstSurface, hipSurfaceObject_t srcSurface, s32 start_x, s32 start_y,
                             s32 end_x, s32 end_y, s64* round, Blizzard* blizzards, u32 blizzard_count, bool backtrack);

__global__ void Clear(hipSurfaceObject_t dstSurface) {
    const Coord block_idx{blockIdx.x, blockIdx.y};
    const Coord block_dim{blockDim.x, blockDim.y};
    const Coord thread_idx{threadIdx.x, threadIdx.y};
    const Coord coord = block_idx.cwiseProduct(block_dim) + thread_idx;

    const char c = Load<char>(dstSurface, coord);
    if (c == 'E') Store(dstSurface, coord, '.');
}

__global__ void MoveBlizzards(hipSurfaceObject_t dstSurface, hipSurfaceObject_t srcSurface, s32 start_x, s32 start_y,
                              s32 end_x, s32 end_y, s64* round, Blizzard* blizzards, s32 blizzard_count, bool backtrack,
                              dim3 grid_size, dim3 block_size) {
    const auto blizzard_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (blizzard_idx >= blizzard_count) return;
    auto& blizzard = blizzards[blizzard_idx];
    switch (blizzard.direction) {
        case '>': {
            if (++blizzard.coord[0] > end_x) blizzard.coord[0] = 1;
        } break;
        case 'v': {
            if (++blizzard.coord[1] >= end_y) blizzard.coord[1] = 1;
        } break;
        case '<': {
            if (--blizzard.coord[0] < 1) blizzard.coord[0] = end_x;
        } break;
        case '^': {
            if (--blizzard.coord[1] < 1) blizzard.coord[1] = end_y - 1;
        } break;
        default: break;
    }
    Store(dstSurface, blizzard.coord, blizzard.direction);

    if (blizzard_idx == 0) {
        ExpandKernel<<<grid_size, block_size>>>(srcSurface, dstSurface, start_x, start_y, end_x, end_y, round,
                                                blizzards, blizzard_count, backtrack);
    }
}

__global__ void ExpandKernel(hipSurfaceObject_t dstSurface, hipSurfaceObject_t srcSurface, s32 start_x, s32 start_y,
                             s32 end_x, s32 end_y, s64* round, Blizzard* blizzards, u32 blizzard_count,
                             bool backtrack) {
    const Coord block_idx{blockIdx.x, blockIdx.y};
    const Coord block_dim{blockDim.x, blockDim.y};
    const Coord thread_idx{threadIdx.x, threadIdx.y};
    const Coord coord = block_idx.cwiseProduct(block_dim) + thread_idx;
    Coord       start_coord{start_x, start_y};
    Coord       end_coord{end_x, end_y};
    if (backtrack) Swap(start_coord, end_coord);

    char c = Load<char>(srcSurface, coord);
    if (c == 0 || c == '#') return;
    if (coord == start_coord) {
        c = 'E';
    }
    for (const auto& vec : {
             Coord{0, -1},
             Coord{-1, 0},
             Coord{1, 0},
             Coord{0, 1},
         }) {
        if (Load<char>(srcSurface, coord + vec) == 'E') {
            c = 'E';
        }
    }
    Store(dstSurface, coord, c);
    if (coord == end_coord) {
        ++(*round);
        if (c != 'E') {
            MoveBlizzards<<<DivCeil(blizzard_count, 1024u), 1024u>>>(dstSurface, srcSurface, start_x, start_y, end_x,
                                                                     end_y, round, blizzards, blizzard_count, backtrack,
                                                                     gridDim, blockDim);
        } else {
            Clear<<<gridDim, blockDim>>>(dstSurface);
            Clear<<<gridDim, blockDim>>>(srcSurface);
        }
    }
}

struct BlizzardMap {
    s32                   width;
    s32                   stride;
    s32                   height;
    std::vector<Blizzard> h_blizzards;
    std::string           h_map;

    UniqueCudaBuffer<s64>      d_round;
    UniqueCudaBuffer<Blizzard> d_blizzards;
    UniqueCudaArray            d_map_src;
    UniqueCudaArray            d_map_dst;
    hipSurfaceObject_t        d_map_src_surface;
    hipSurfaceObject_t        d_map_dst_surface;

    Coord beginCoord() const { return {1, 0}; }
    Coord endCoord() const { return {width - 2, height - 1}; }

    BlizzardMap(std::istream& is) {
        h_map  = std::string{std::istreambuf_iterator<char>{is}, {}};
        width  = h_map.find('\n');
        stride = width + 1;
        height = h_map.size() / stride;

        for (s32 y = 1; y < height - 1; ++y) {
            for (s32 x = 1; x < width - 1; ++x) {
                auto& cell = h_map[y * stride + x];
                if (cell == '>' || cell == 'v' || cell == '<' || cell == '^') {
                    h_blizzards.emplace_back(Blizzard{
                        Coord{x, y},
                        cell,
                    });
                    cell = '.';
                }
            }
        }

        d_round = MakeDeviceBuffer<s64>(1);
        Check(hipMemset(d_round.get(), 0, sizeof(s64)));
        d_blizzards = MakeDeviceBuffer<Blizzard>(h_blizzards.size());
        Check(hipMemcpy(d_blizzards.get(), h_blizzards.data(), h_blizzards.size() * sizeof(Blizzard),
                         hipMemcpyHostToDevice));
        const auto d_map_format = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindSigned);
        d_map_src               = MakeDeviceArray(d_map_format, width, height, hipArraySurfaceLoadStore);
        d_map_dst               = MakeDeviceArray(d_map_format, width, height, hipArraySurfaceLoadStore);

        Check(hipMemcpy2DToArray(d_map_src.get(), 0, 0, h_map.data(), stride, width, height, hipMemcpyHostToDevice));
        Check(hipMemcpy2DToArray(d_map_dst.get(), 0, 0, h_map.data(), stride, width, height, hipMemcpyHostToDevice));

        hipResourceDesc descriptor{};
        descriptor.resType = hipResourceTypeArray;

        descriptor.res.array.array = d_map_src.get();
        Check(hipCreateSurfaceObject(&d_map_src_surface, &descriptor));
        descriptor.res.array.array = d_map_dst.get();
        Check(hipCreateSurfaceObject(&d_map_dst_surface, &descriptor));
    };

    char& Cell(std::string& map, const Coord& coord) {
        assert(coord[0] >= 0 && coord[0] < width && coord[1] >= 0 && coord[1] < height);
        return map[coord[1] * stride + coord[0]];
    }

    ~BlizzardMap() {
        Check(hipDestroySurfaceObject(d_map_src_surface));
        Check(hipDestroySurfaceObject(d_map_dst_surface));
    }

    bool advance(bool backtrack) {
        auto new_map = h_map;
        for (s32 y = 1; y < height - 1; ++y) {
            for (s32 x = 1; x < width - 1; ++x) {
                Coord coord{x, y};
                for (const auto& vec : {
                         Coord{0, 0},
                         Coord{0, -1},
                         Coord{-1, 0},
                         Coord{1, 0},
                         Coord{0, 1},
                     }) {
                    if (Cell(h_map, coord + vec) == 'E') {
                        Cell(new_map, coord) = 'E';
                    }
                }
            }
        }
        if (!backtrack || Cell(h_map, {1, 1}) == 'E') {
            Cell(new_map, beginCoord()) = 'E';
        }
        if (backtrack || Cell(h_map, {width - 2, height - 2}) == 'E') {
            Cell(new_map, endCoord()) = 'E';
        }
        h_map = std::move(new_map);

        for (Blizzard& blizzard : h_blizzards) {
            switch (blizzard.direction) {
                case '>': {
                    if (++blizzard.coord[0] >= width - 1) blizzard.coord[0] = 1;
                } break;
                case 'v': {
                    if (++blizzard.coord[1] >= height - 1) blizzard.coord[1] = 1;
                } break;
                case '<': {
                    if (--blizzard.coord[0] < 1) blizzard.coord[0] = width - 2;
                } break;
                case '^': {
                    if (--blizzard.coord[1] < 1) blizzard.coord[1] = height - 2;
                } break;
                default: assert(false); break;
            }
            Cell(h_map, blizzard.coord) = blizzard.direction;
        }
        // fmt::print("{}\n\n", map);
        if (backtrack) {
            return Cell(h_map, beginCoord()) == 'E';
        } else {
            return Cell(h_map, endCoord()) == 'E';
        }
    }

    void clear() {
        for (auto& c : h_map) {
            if (c == 'E') c = '.';
        }
    }

    s64 runCU(bool backtrack) {
        auto start_x = 1;
        auto start_y = 0;
        auto end_x   = width - 2;
        auto end_y   = height - 1;
        dim3 block_dim{
            32,
            32,
        };
        dim3 grid_dim{
            DivCeil<u32>(width, block_dim.x),
            DivCeil<u32>(height, block_dim.y),
        };
        ExpandKernel<<<grid_dim, block_dim>>>(d_map_src_surface, d_map_dst_surface, start_x, start_y, end_x, end_y,
                                              d_round.get(), d_blizzards.get(), h_blizzards.size(), backtrack);
        Check(hipPeekAtLastError());
        s64 round = 0;
        Check(hipMemcpy(&round, d_round.get(), sizeof(round), hipMemcpyDeviceToHost));
        return round;
    }
};

void Run(BlizzardMap& blizzard_map, bool gpu) {
    auto start_time = std::chrono::steady_clock::now();
    s32  part1      = 0;
    s32  part15     = 0;
    s32  part2      = 0;
    if (!gpu) {
        s32 rounds = 0;
        while (!blizzard_map.advance(false))
            ++rounds;
        ++rounds;
        part1 = rounds;
        blizzard_map.clear();
        while (!blizzard_map.advance(true))
            ++rounds;
        ++rounds;
        part15 = rounds;
        blizzard_map.clear();
        while (!blizzard_map.advance(false))
            ++rounds;
        ++rounds;
        part2 = rounds;
    } else {
        part1  = blizzard_map.runCU(false);
        part15 = blizzard_map.runCU(true) - 1;
        part2  = blizzard_map.runCU(false) - 2;
    }
    auto end_time = std::chrono::steady_clock::now();

    fmt::print("\n---- {} ----\n", gpu ? "GPU" : "CPU");
    fmt::print("Part   1: {} rounds\n", part1);
    fmt::print("Part 1.5: {} rounds\n", part15);
    fmt::print("Part   2: {} rounds\n", part2);
    fmt::print("Simulation Time: {}\n", end_time - start_time);
}

int main() {
    std::ifstream input_file{"input.txt"};
    BlizzardMap   blizzard_map{input_file};
    Run(blizzard_map, false);
    Run(blizzard_map, true);
}