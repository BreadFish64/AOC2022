#include "hip/hip_runtime.h"
#include "pch.hpp"

#include "monkey_math.inl"

std::istringstream test{R"(root: pppw + sjmn
dbpl: 5
cczh: sllz + lgvd
zczc: 2
ptdq: humn - dvpt
dvpt: 3
lfqf: 4
humn: 5
ljgn: 2
sjmn: drzm * dbpl
sllz: 4
pppw: cczh / lfqf
lgvd: ljgn * ptdq
drzm: hmdt - zczc
hmdt: 32
)"};

constexpr s32 Id(std::string_view sv) {
    assert(sv.size() == 4);
    s32 id = 0;
    for (char c : sv) {
        id *= 26;
        id += c - 'a';
    }
    return id;
}

struct Monkey {
    s32  lhs;
    s32  rhs;
    char op;
};

s64 Solve(std::span<const Monkey, Power<usize>(26, 4)> monkeys, s32 id) {
    const auto monkey = monkeys[id];
    switch (monkey.op) {
        case '=': return monkey.lhs;
        case '+': return Solve(monkeys, monkey.lhs) + Solve(monkeys, monkey.rhs);
        case '-': return Solve(monkeys, monkey.lhs) - Solve(monkeys, monkey.rhs);
        case '*': return Solve(monkeys, monkey.lhs) * Solve(monkeys, monkey.rhs);
        case '/': return Solve(monkeys, monkey.lhs) / Solve(monkeys, monkey.rhs);
        default: assert(false); return 0;
    }
}

constexpr auto ROOT_ID = Id("root");
constexpr auto HUMN_ID = Id("humn");

constexpr void Part2() {
    constexpr s64                  range = 1_s64 << 53;
    std::ofstream                  solution{"solution.txt"};
    std::vector<std::future<void>> threads;
    for (s64 i = 0; i < 8; ++i) {
        s64 begin = i * (range / 8);
        s64 end   = (i + 1) * (range / 8);
        threads.emplace_back(std::async(
            std::launch::async,
            [&solution](s64 begin, s64 end) {
                for (s64 i = begin; i < end; ++i) {
                    if ((i & ((1_s64 << 32) - 1)) == 0) {
                        fmt::print("Progress: {}\n", i);
                    }
                    if (hardcoded_monkeys::root2(i)) {
                        fmt::print("Part 2: {}\n", i);
                        solution << i << '\n';
                        break;
                    }
                }
            },
            begin, end));
        threads.emplace_back(std::async(
            std::launch::async,
            [&solution](s64 begin, s64 end) {
                for (s64 i = begin; i > end; --i) {
                    if ((i & ((1_s64 << 32) - 1)) == 0) {
                        fmt::print("Progress: {}\n", i);
                    }
                    if (hardcoded_monkeys::root2(i)) {
                        fmt::print("Part 2: {}\n", i);
                        solution << i << '\n';
                        break;
                    }
                }
            },
            -begin, -end));
    }
    for (auto& thread : threads)
        thread.wait();
}

__global__ Part2_Kernel(s64 offset, s64* result) {
    s64 num = static_cast<s64>(blockDim.x) * static_cast<s64>(blockIdx.x) + static_cast<s64>(threadIdx.x) + offset;
    if (root2(num)) *result = num;
}

constexpr void CUPart2() {
    constexpr s64 range      = 1_s64 << 53;
    constexpr s64 block_size = 1024;
    constexpr s64 grid_size  = 1_s64 << 30;
    constexpr s64 thread_count  = grid_size * block_size;
    s64           result        = 0;
    auto          result_buffer = MakeDeviceBuffer<s64>(1);
    Check(hipMemcpy(result_buffer.get(), &result, sizeof(result), hipMemcpyHostToDevice));

    for (s64 i = -range + 1; i < range; i += thread_count) {
        fmt::print("{}\n", i);
        Part2_Kernel<<<grid_size, block_size>>>(i, result_buffer.get());
        Check(hipMemcpy(&result, result_buffer.get(), sizeof(result), hipMemcpyDeviceToHost));
        if (result != 0) {
            fmt::print("Part 2: {}\n", result);
        }
    }
}

int main() {
    std::ifstream input_file{"input.txt"};
    std::istream& input = input_file;
    std::string   line_buffer;

    std::vector<Monkey>                    monkey_buffer(Power(26, 4));
    std::span<Monkey, Power<usize>(26, 4)> monkeys{monkey_buffer};

    while (std::getline(input, line_buffer)) {
        std::string_view line{line_buffer};
        Monkey&          monkey = monkeys[Id(line.substr(0, 4))];
        if (isdigit(line[6])) {
            std::from_chars(line.data() + 6, line.data() + line.size(), monkey.lhs);
            monkey.op = '=';
        } else {
            monkey.lhs = Id(line.substr(6, 4));
            monkey.op  = line[11];
            monkey.rhs = Id(line.substr(13, 4));
        }
    }
    fmt::print("Part 1: {}\n", Solve(monkeys, ROOT_ID));
    fmt::print("Part 1: {}\n", hardcoded_monkeys::root(855));

    Part2();
}
