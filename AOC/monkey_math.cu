#include "hip/hip_runtime.h"
#include "pch.hpp"
#include "cuda.hpp"

#include "monkey_math.inl"


void Part2() {
    constexpr s64                  range = 1_s64 << 53;
    std::ofstream                  solution{"solution.txt"};
    std::vector<std::future<void>> threads;
    for (s64 i = 0; i < 8; ++i) {
        s64 begin = i * (range / 8);
        s64 end   = (i + 1) * (range / 8);
        threads.emplace_back(std::async(
            std::launch::async,
            [&solution](s64 begin, s64 end) {
                for (s64 i = begin; i < end; ++i) {
                    if ((i & ((1_s64 << 32) - 1)) == 0) {
                        fmt::print("Progress: {}\n", i);
                    }
                    if (hardcoded_monkeys::root2(i)) {
                        fmt::print("Part 2: {}\n", i);
                        solution << i << '\n';
                        break;
                    }
                }
            },
            begin, end));
        threads.emplace_back(std::async(
            std::launch::async,
            [&solution](s64 begin, s64 end) {
                for (s64 i = begin; i > end; --i) {
                    if ((i & ((1_s64 << 32) - 1)) == 0) {
                        fmt::print("Progress: {}\n", i);
                    }
                    if (hardcoded_monkeys::root2(i)) {
                        fmt::print("Part 2: {}\n", i);
                        solution << i << '\n';
                        break;
                    }
                }
            },
            -begin, -end));
    }
    for (auto& thread : threads)
        thread.wait();
}

__global__ void Part2_Kernel(s64 offset, s64* result) {
    s64 num = static_cast<s64>(blockDim.x) * static_cast<s64>(blockIdx.x) + static_cast<s64>(threadIdx.x) + offset;
    if (hardcoded_monkeys::root2(num)) *result = num;
}

void CUPart3() {
    // checked -range to -8957721231491072
    constexpr s64 range      = 1_s64 << 53;
    constexpr s64 block_size = 1024;
    constexpr s64 grid_size  = 1_s64 << 11;
    constexpr s64 thread_count  = grid_size * block_size;
    constexpr s64 start         = 3412619952128;
    s64           result        = 0;
    auto          result_buffer = MakeDeviceBuffer<s64>(1);
    Check(hipMemcpy(result_buffer.get(), &result, sizeof(result), hipMemcpyHostToDevice));

    for (s64 i = start; i < range; i += thread_count) {
        fmt::print("{}\n", i);
        std::fflush(stdout);
        Part2_Kernel<<<grid_size, block_size>>>(i, result_buffer.get());
        Check(hipMemcpy(&result, result_buffer.get(), sizeof(result), hipMemcpyDeviceToHost));
        if (result != 0) {
            fmt::print("Part 2: {}\n", result);
            std::fflush(stdout);
            break;
        }
    }
}

void CUPart2() {
    // checked -range to -8957721231491072
    constexpr s64 range         = 1_s64 << 53;
    constexpr s64 block_size    = 1024;
    constexpr s64 grid_size     = 1_s64 << 30;
    constexpr s64 thread_count  = grid_size * block_size;
    s64           result        = 0;
    auto          result_buffer = MakeDeviceBuffer<s64>(1);
    Check(hipMemcpy(result_buffer.get(), &result, sizeof(result), hipMemcpyHostToDevice));

    for (s64 i = -thread_count; i >= -range; i -= thread_count) {
        fmt::print("{}\n", i);
        std::fflush(stdout);
        Part2_Kernel<<<grid_size, block_size>>>(i, result_buffer.get());
        Check(hipMemcpy(&result, result_buffer.get(), sizeof(result), hipMemcpyDeviceToHost));
        if (result != 0) {
            fmt::print("Part 2: {}\n", result);
            std::fflush(stdout);
            break;
        }
    }
}

int main() {
    fmt::print("Part 1: {}\n", hardcoded_monkeys::root(855));
    fmt::print("Part 2: {}\n", hardcoded_monkeys::root2(3412650897408_s64));
    //CUPart3();

    for (s64 i = 3412649312256_s64 ; i < (1_s64 << 53); ++i) {
        if (hardcoded_monkeys::root2(i)) {
            fmt::print("Part 2: {}\n", i);
            std::fflush(stdout);
            break;
        }
    }
}
