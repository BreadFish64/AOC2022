#include "hip/hip_runtime.h"
#include "pch.hpp"

#include "cuda.hpp"

#include <gsl/gsl>

std::istringstream test{R"(..............
..............
.......#......
.....###.#....
...#...#.#....
....#...##....
...#.###......
...##.#.##....
....#..#......
..............
..............
..............
)"};

using Coord = Eigen::Vector2i;

enum Direction : s8 {
    NORTH = 8,
    SOUTH,
    WEST,
    EAST,
    END,
    NO_MOVE = 127,
    NONE    = 0,
};

__device__ constexpr s32 GRID_SIZE   = 156;
__device__ constexpr s32 GRID_CENTER = 63;

__device__ constexpr s32 BLOCK_SIZE    = 32;
__device__ constexpr s32 BOUNDARY_SIZE = 3;
__device__ constexpr s32 TILE_SIZE     = BLOCK_SIZE - (2 * BOUNDARY_SIZE);

__global__ void ElfMoveKernel(hipSurfaceObject_t elf_surface, s8 proposed_direction, s64* dirty) {
    const std::array<Coord, 4> DIR_LUT{{
        {0, -1},
        {0, 1},
        {-1, 0},
        {1, 0},
    }};

    const Eigen::Vector2i block_size{BLOCK_SIZE, BLOCK_SIZE};
    const Eigen::Vector2i tile_size{TILE_SIZE, TILE_SIZE};
    const Eigen::Vector2i boundary_size{BOUNDARY_SIZE, BOUNDARY_SIZE};

    const Eigen::Vector2i tile_idx{blockIdx.x, blockIdx.y};
    const Eigen::Vector2i thread_idx{threadIdx.x, threadIdx.y};
    const Eigen::Vector2i tile_offset = thread_idx - boundary_size;
    const Eigen::Vector2i global_idx  = tile_idx.cwiseProduct(tile_size) + tile_offset;

    __shared__ s8 elves[BLOCK_SIZE][BLOCK_SIZE];
    const auto    GetElf = [&](Coord idx) -> s8& {
        return elves[idx[1]][idx[0]];
    };
    __shared__ bool proposed_moves[BLOCK_SIZE][BLOCK_SIZE][2];
    proposed_moves[thread_idx[1]][thread_idx[0]][0] = false;
    proposed_moves[thread_idx[1]][thread_idx[0]][1] = false;

    auto& cc = GetElf(thread_idx);
    cc       = Load<s8>(elf_surface, global_idx);

    __syncthreads();

    bool dead_thread =
        thread_idx[0] == 0 || thread_idx[0] == BLOCK_SIZE - 1 || thread_idx[1] == 0 || thread_idx[1] == BLOCK_SIZE - 1;

    auto proposed_coord = thread_idx;
    if (!dead_thread && cc != NONE) {
        u8 empty_spots = 0;
        empty_spots |= GetElf(thread_idx + Coord{-1, -1}) == NONE ? 0x01 : 0;
        empty_spots |= GetElf(thread_idx + Coord{0, -1}) == NONE ? 0x02 : 0;
        empty_spots |= GetElf(thread_idx + Coord{1, -1}) == NONE ? 0x04 : 0;
        empty_spots |= GetElf(thread_idx + Coord{-1, 0}) == NONE ? 0x08 : 0;
        empty_spots |= GetElf(thread_idx + Coord{1, 0}) == NONE ? 0x10 : 0;
        empty_spots |= GetElf(thread_idx + Coord{-1, 1}) == NONE ? 0x20 : 0;
        empty_spots |= GetElf(thread_idx + Coord{0, 1}) == NONE ? 0x40 : 0;
        empty_spots |= GetElf(thread_idx + Coord{1, 1}) == NONE ? 0x80 : 0;

        u8 valid_directions = 0;
        valid_directions |= (empty_spots & 0b00000111) == 0b00000111 ? 0x01 : 0;
        valid_directions |= (empty_spots & 0b11100000) == 0b11100000 ? 0x02 : 0;
        valid_directions |= (empty_spots & 0b00101001) == 0b00101001 ? 0x04 : 0;
        valid_directions |= (empty_spots & 0b10010100) == 0b10010100 ? 0x08 : 0;

        cc = NO_MOVE;
        if (valid_directions != 0 && valid_directions != 0xF) {
            while (true) {
                if (valid_directions & (1 << (proposed_direction - NORTH))) {
                    cc = proposed_direction;
                    proposed_coord += DIR_LUT[proposed_direction - NORTH];
                    proposed_moves[proposed_coord[1]][proposed_coord[0]][proposed_direction & 1] = true;
                    break;
                }
                if (++proposed_direction == END) proposed_direction = NORTH;
            }
        }
    }

    auto& is_proposed = proposed_moves[proposed_coord[1]][proposed_coord[0]];

    __syncthreads();

    if (!dead_thread && (cc >= NORTH && cc < END)) {
        if (is_proposed[proposed_direction & 1] && !is_proposed[(proposed_direction & 1) ^ 1]) {
            *dirty                 = true;
            cc                     = NONE;
            GetElf(proposed_coord) = NO_MOVE;
        }
    }

    __syncthreads();

    dead_thread |=
        tile_offset[0] < 0 || tile_offset[0] >= TILE_SIZE || tile_offset[1] < 0 || tile_offset[1] >= TILE_SIZE;
    if (dead_thread) return;

    Store(elf_surface, global_idx, cc);
}

int main() {
    std::ifstream input_file{"input.txt"};
    std::string   input{std::istreambuf_iterator<char>{input_file}, {}};

    auto       d_elf_dirty_buffer = MakeDeviceBuffer<s64>(1);
    const auto elf_format         = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindSigned);
    auto       d_elf_buffer       = MakeDeviceArray(elf_format, GRID_SIZE, GRID_SIZE, hipArraySurfaceLoadStore);

    auto      start_time = std::chrono::steady_clock::now();
    const s32 width      = input.find('\n');
    const s32 stride     = width + 1;
    const s32 height     = input.size() / stride;

    std::vector<std::array<s8, GRID_SIZE>> elves(GRID_SIZE);
    for (auto& row : elves)
        std::fill(row.begin(), row.end(), NONE);

    for (s32 y = 0; y < height; ++y) {
        for (s32 x = 0; x < width; ++x) {
            if (input[y * stride + x] == '#')
                elves[y + GRID_CENTER - (height / 2)][x + GRID_CENTER - (width / 2)] = NO_MOVE;
        }
    }

    Check(hipMemcpy2DToArray(d_elf_buffer.get(), 0, 0, elves.data(), GRID_SIZE, GRID_SIZE, GRID_SIZE,
                              hipMemcpyHostToDevice));

    const auto PrintElves = [&elves, &d_elf_buffer] {
        for (const auto& row : elves) {
            for (auto e : row) {
                fmt::print("{}", e == NONE ? '.' : '#');
            }
            fmt::print("\n");
        }
        fmt::print("\n");
    };

    hipResourceDesc descriptor{};
    descriptor.resType         = hipResourceTypeArray;
    descriptor.res.array.array = d_elf_buffer.get();

    hipSurfaceObject_t elf_surface{};
    Check(hipCreateSurfaceObject(&elf_surface, &descriptor));
    auto cleanup_surfaces = gsl::finally([&] { Check(hipDestroySurfaceObject(elf_surface)); });

    s64        elves_moved     = true;
    s8         round_direction = NORTH;
    s32        round_count     = 0;
    const auto DoRound         = [&] {
        ++round_count;
        elves_moved = false;

        Check(hipMemcpy(d_elf_dirty_buffer.get(), &elves_moved, sizeof(elves_moved), hipMemcpyHostToDevice));
        ElfMoveKernel<<<dim3{
                            DivCeil(GRID_SIZE, TILE_SIZE),
                            DivCeil(GRID_SIZE, TILE_SIZE),
                        },
                        dim3{
                            BLOCK_SIZE,
                            BLOCK_SIZE,
                        }>>>(elf_surface, round_direction, d_elf_dirty_buffer.get());
        Check(hipPeekAtLastError());
        Check(hipMemcpy(&elves_moved, d_elf_dirty_buffer.get(), sizeof(elves_moved), hipMemcpyDeviceToHost));

        if (++round_direction == END) round_direction = NORTH;
    };
    while (round_count < 10)
        DoRound();
    Check(hipMemcpy2DFromArray(elves.data(), GRID_SIZE, d_elf_buffer.get(), 0, 0, GRID_SIZE, GRID_SIZE,
                                hipMemcpyDeviceToHost));

    s32 elf_count = 0;
    s32 min_x = GRID_SIZE, min_y = GRID_SIZE, max_x = 0, max_y = 0;
    for (s32 y = 0; y < GRID_SIZE; ++y) {
        for (s32 x = 0; x < GRID_SIZE; ++x) {
            if (elves[y][x] != NONE) {
                ++elf_count;
                min_x = std::min(min_x, x);
                min_y = std::min(min_y, y);
                max_x = std::max(max_x, x);
                max_y = std::max(max_y, y);
            }
        }
    }
    const s32 empty_cells = (max_y - min_y + 1) * (max_x - min_x + 1) - elf_count;

    while (elves_moved)
        DoRound();

    auto end_time = std::chrono::steady_clock::now();

    fmt::print("\nBounding Box: y = [{}, {}], x = [{}, {}]\n", min_y, max_y, min_x, max_x);
    fmt::print("Elves: {}\nEmpty Cells: {}\n", elf_count, empty_cells);

    fmt::print("Elves stopped after {} rounds\n", round_count);
    fmt::print("Time: {}\n", end_time - start_time);
}